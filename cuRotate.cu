#include "hip/hip_runtime.h"

#include "../common/book.h"
#include "./utils.h"
#include "./timing.h"
#include "./itk_io.h"
// #include "./cuda_kernels.cuh"
#pragma clang diagnostic push
#pragma ide diagnostic ignored "CannotResolve"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <complex>
#include <vector>
#include <cstring>


#define NX 128
#define NY 128
#define NZ 128
#define LX (2 * M_PI)
#define LY (2 * M_PI)
#define NUM_IMAGES 1
#define THETA 3*M_PI_2
#define PHI 0


// #define ST sin(THETA)
// #define CT cos(THETA)
// #define SP sin(PHI)
// #define CP cos(PHI)

texture<float, 3, hipReadModeElementType> tex;

using namespace std;

typedef float     SimPixelType;

// __global__ void add_slices(PIXEL_TYPE* image_in, PIXEL_TYPE* image_out) {
//     int tid = threadIdx.x + blockIdx.x * blockDim.x;
//     int idx = tid % (128 * 128 * 2);
//     PIXEL_TYPE temp = image_in[tid];
//     // printf("%d\n", idx);
//     // if (tid < TOTAL_PIXELS) {
//     // if (tid < 16384) {
//     atomicAdd( &image_out[idx], temp );
//         // image_out[tid] = temp;
//     // }
//     // }
// }

/*
*	Texture lookup based 3D volume rotation.
*	
*
*/
__global__ void
d_render(float *d_output /*, uint imageW, uint imageH, float w*/)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    int z = tid / 16384 + 1;
    int x = tid % 128 + 1;
    int y = ( tid % 16384 ) / 128 + 1;

	float ST = sinf(THETA);
	float CT = cosf(THETA);
	float SP = sinf(PHI);
	float CP = cosf(PHI);

    int p1 = (NX + 1)/2 + 1;
    int p2 = (NY + 1)/2 + 1;
    int p3 = (NZ + 1)/2 + 1;

    // Apply the rotation, nearest neighbor
    float xx = roundf(x*CT + z*ST - CT*p1 + p1 - ST*p3);
    float yy = roundf(- x*SP*ST + y*CP + z*SP*CT + SP*ST*p1 - CP*p2 - SP*CT*p3 + p2);
    float zz = roundf(- x*CP*ST - y*SP + z*CP*CT + CP*ST*p1 + SP*p2 - CP*CT*p3 + p3);
    if (xx <= NX && xx >= 1 && yy <= NY && yy >= 1 && zz <= NZ && zz >= 1) {
    	uint idx = (zz - 1) * 16384 + (yy - 1) * 128 + xx - 1;
    	float voxel = tex3D( tex, x - 1, y - 1, z - 1 );
    	d_output[idx] = voxel;
    }

    // Apply a texture lookup
    // d_output[tid] = voxel;
    // atomicAdd( &d_output[tid], voxel );
}

 __global__ void Multiply_complex(SimPixelType* image_in, SimPixelType* image_in2) {
     int tid = threadIdx.x + blockIdx.x * blockDim.x;
//     int idx = tid % (128 * 128 * 2);
     SimPixelType c1_real = image_in[tid*2];
	 SimPixelType c1_imag = image_in[tid*2+1];
	 SimPixelType c2_real = image_in2[tid*2];
	 SimPixelType c2_imag = image_in2[tid*2+1];
	 image_in[tid*2] = c1_real * c2_real - c1_imag * c2_imag;
	 image_in[tid*2+1] = c1_real * c2_imag + c1_imag * c2_real;
 }

int main() {
	hipDeviceReset();
	/* Create couple of images for testing */
	SimPixelType *x = new SimPixelType[NX * NY];
	SimPixelType *y = new SimPixelType[NX * NY];
	SimPixelType *in = new SimPixelType[NX * NY * NZ];
	/* A vector holding multiple images data */
	vector< SimPixelType* > image_vector;
	vector< SimPixelType* > dev_pointers_in;
	vector< SimPixelType* > dev_pointers_out;
	vector< SimPixelType* > imageOut_vector;
	vector< SimPixelType* > mult_image_vector;

	/* Create Fourier Kernel plan */
	hipfftHandle planr2c[NUM_IMAGES];
	hipfftHandle planc2r[NUM_IMAGES];

	/* Create an array of CUDA streams */
	hipStream_t streams_fft[NUM_IMAGES];

	/* Output image */
	complex<SimPixelType> *out = new complex<SimPixelType>[NX * NY * NZ];
	gpuErrchk( hipHostRegister( out, sizeof(SimPixelType)*NX*NY*NZ*2, hipHostRegisterPortable ) );
	// complex<SimPixelType>* out;
	// gpuErrchk( hipHostMalloc( &out, NX * NY * NZ * sizeof(SimPixelType) * 2 ) );
		/* Initialize it */
	memset( out, 0, sizeof(SimPixelType)*NX*NY*NZ*2 );

	/* Create the second argument image in the multiply kernel */
	SimPixelType* OTF = new SimPixelType[NX * NY * NZ * 2]; // Since the image is complex
	SimPixelType* dev_OTF;

	for (int p = 0; p < NZ; p++) {
		for(int j = 0; j < NY; j++) {
			for(int kk = 0; kk < NX; kk++) {
				OTF[(j * NX + kk) * 2] = kk + j;
				OTF[(j * NX + kk) * 2 + 1] = kk + j;
			}
		}
	}
	/* Reserve memory locations for the OTF image */
	gpuErrchk( hipMalloc( &dev_OTF, sizeof(SimPixelType)*NX*NY*NZ*2 ) );
	gpuErrchk( hipHostRegister( OTF, sizeof(SimPixelType)*NX*NY*NZ*2, hipHostRegisterPortable ) );

	for (unsigned i = 0; i < NUM_IMAGES; i++) {

		SimPixelType *vx = new SimPixelType[NX * NY * NZ];
		SimPixelType *mult_image = new SimPixelType[NX * NY * NZ];
		// SimPixelType* vx;
		// hipHostMalloc( &vx, NX * NY * NZ * sizeof(SimPixelType) );
		for (int p = 0; p < NZ; p++) {
			for(int j = 0; j < NY; j++){
			    for(int kk = 0; kk < NX; kk++){
			        x[j * NX + kk] = kk * LX/NX;
			        y[j * NX + kk] = kk * LY/NY;

			        /* Put values in the new images */
			        vx[j * NX + kk + p * NX * NY] = cos(x[j * NX + kk] + y[j * NX + kk]);
			        if ( i == 0 ) {
			        	in[j * NX + kk + p * NX * NY] = cos(x[j * NX + kk] + y[j * NX + kk]);
			        }
			    }
			}
		}
		t1 = absoluteTime();
		gpuErrchk( hipHostRegister( vx, sizeof(SimPixelType)*NX*NY*NZ, hipHostRegisterPortable ) );
		gpuErrchk( hipHostRegister( mult_image, sizeof(SimPixelType)*NX*NY*NZ, hipHostRegisterPortable ) );
		t2 = absoluteTime();
  		std::cout << "\n\n Register time: " << (float)(t2-t1)/1000000 << "ms" << std::endl;
		// for (int j = 0; j < NY; j++){
		//     for (int i = 0; i < NX; i++){
		//         // printf("%.3f ", vx[j*NX + i]/(NX*NY));
		//         cout << vx[j * NX + i] << " ";
		//     }
		//     // printf("\n");
		//     cout << endl;
		// }
		// cout << endl;
		/* Allocate some spaces on the device */
		SimPixelType *d_vx;
		SimPixelType *d_out;
		/* Some space on the device */
		gpuErrchk(hipMalloc(&d_vx, NX * NY * NZ * sizeof(SimPixelType)));
		gpuErrchk(hipMalloc(&d_out, NX * NY * NZ * sizeof(hipfftReal)));

		/* Create cufft FFT plans */
		int n[2] = {NX, NY};
		int inembed[] = {NX, NY};
		int onembed[] = {NX, NY};

		/* Forward Fourier Transform plan */
		hipfftPlanMany(&planr2c[i],
		            2, // rank
		            n, // dimension
		            inembed,
		            1, // istride
		            NX * NY, // idist
		            onembed,
		            1, //ostride
		            NX * NY, // odist
		            HIPFFT_R2C,
		            NZ);



		/* Inverse Fourier Transform plan */
		hipfftPlanMany(&planc2r[i],
		            2, // rank
		            n, // dimension
		            onembed,
		            1, // istride
		            NX * NY, // idist
		            inembed,
		            1, //ostride
		            NX * NY, // odist
		            HIPFFT_C2R,
		            NZ);

		cufftSetCompatibilityMode(planr2c[i], CUFFT_COMPATIBILITY_NATIVE);
		cufftSetCompatibilityMode(planc2r[i], CUFFT_COMPATIBILITY_NATIVE);
		/* Create streams associated with this 2 plans  */
		gpuErrchk( hipStreamCreate( &streams_fft[i] ));
		hipfftSetStream( planr2c[i], streams_fft[i] );
		// gpuErrchk( hipStreamCreate(&streams_ifft[i]) );
		// hipfftSetStream(&planc2r[i]);

		image_vector.push_back( vx );
		mult_image_vector.push_back( mult_image );
		dev_pointers_in.push_back( d_vx );
		dev_pointers_out.push_back( d_out );
	}


	/* Copying data to the device for processing */
	// hipMemcpy(d_vx, vx, NX * NY * sizeof(hipfftDoubleReal), hipMemcpyHostToDevice);
	// hipMemcpy(d_out, out, NX * NY * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice);

	gpuErrchk( hipMemcpyAsync(
				dev_OTF,
				OTF,
				2*NX*NY*NZ*sizeof(SimPixelType),
				hipMemcpyHostToDevice,
				streams_fft[0]
	) );

	for (unsigned int j = 0; j < NUM_IMAGES; j++ ) {
		gpuErrchk( hipMemcpyAsync( dev_pointers_in[j],
									image_vector[j],
									NX*NY*NZ*sizeof(SimPixelType),
									hipMemcpyHostToDevice,
									streams_fft[j]) );
		gpuErrchk( hipMemcpyAsync( dev_pointers_out[j],
									out,
									NX*NY*NZ*sizeof(hipfftReal),
									hipMemcpyHostToDevice,
									streams_fft[j] ) );

	}
	/*
	*	Apply the rotation
	*/
	/* Create texture array */
	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		gpuErrchk( hipStreamSynchronize( streams_fft[j] ) );
	}
	t1 = absoluteTime();
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray *d_volumeArray = 0;
	const hipExtent volumeSize = make_hipExtent(128, 128, 128);
	size_t size = volumeSize.width*volumeSize.height*volumeSize.depth;
	gpuErrchk( hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize) );
	hipMemcpy3DParms copyParams = {0};

	copyParams.srcPtr = make_hipPitchedPtr( dev_pointers_in[0],
											 volumeSize.width*sizeof(float),
											 volumeSize.width,
											 volumeSize.height
											  );
	copyParams.dstArray = d_volumeArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyDeviceToDevice;
	gpuErrchk( hipMemcpy3D( &copyParams) );

	tex.normalized = false;
	tex.filterMode = hipFilterModePoint; // Filtering mode
	tex.addressMode[0] = hipAddressModeBorder;
	tex.addressMode[1] = hipAddressModeBorder;
	tex.addressMode[2] = hipAddressModeBorder;
	gpuErrchk(hipBindTextureToArray(tex, d_volumeArray, channelDesc));

	// Obtain data from texture memory
	d_render<<< NX*NY*NZ/256, 256, 0, streams_fft[0] >>>( dev_pointers_out[0] );

	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		// hipfftExecD2Z( planr2c[j],
		// 			  (SimPixelType*)dev_pointers_in[j],
		// 			  (hipfftDoubleComplex*)dev_pointers_out[j]);
		// Multiply_complex<<< NX*NY*NZ/512, 512, 0, streams_fft[j] >>>( dev_pointers_out[j],
		// 				  dev_OTF
		// 					);

		/* CUDA rotation */
	}


//	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
//		hipfftSetStream(planc2r[j], streams_fft[j]);
//	}
//
//	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
//		hipfftExecZ2D( planc2r[j], (hipfftDoubleComplex*)dev_pointers_out[j], (SimPixelType*)dev_pointers_in[j]);
//	}

	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		gpuErrchk( hipMemcpyAsync( mult_image_vector[j], dev_pointers_out[j], NX*NY*NZ*sizeof(SimPixelType), hipMemcpyDeviceToHost, streams_fft[j] ) );
	}

	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		gpuErrchk( hipStreamSynchronize( streams_fft[j] ) );
	}

	t2 = absoluteTime();
  	std::cout << "\n\n Streaming time: " << (float)(t2-t1)/1000000 << "ms" << std::endl;
 	t1 = absoluteTime();
	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		gpuErrchk( hipHostUnregister(image_vector[j]) );
		// gpuErrchk( hipHostFree(image_vector[j]) );
	}
	gpuErrchk( hipHostUnregister(OTF) );
	gpuErrchk( hipHostUnregister(out) );
	// gpuErrchk( hipHostFree( out ) );
	t2 = absoluteTime();
  	std::cout << "\n\n Host Unregister time: " << (float)(t2-t1)/1000000 << "ms" << std::endl;


//    	for (int j = 0; j < NY; j++){
// 	     for (int i = 0; i < NX; i++){
// 	         // printf("%.3f ", vx[j*NX + i]/(NX*NY));
// 	         // SimPixelType* vx = image_vector[1];
// //	         cout << image_vector[0][j * NX + i]/( NX * NY ) << " ";
// 			cout << complex_array[j * NX + i] << " ";
// 	     }
// 	     // printf("\n");
// 	     cout << endl;
// 	 }
	// cout << endl;
	// for (int j = 0; j < NY; j++){
	//     for (int i = 0; i < NX; i++){
	//         // printf("%.3f ", vx[j*NX + i]/(NX*NY));
	//         cout << in[j * NX + i] << " ";
	//     }
	//     // printf("\n");
	//     cout << endl;
	// }

	/*
	*	Output an image for testing
	*/
	store_mha/*<double>*/( mult_image_vector[0],// input image
	 			3, // dim
	 			NX,// h
	 			NY,// w
	 			NZ,// d
	 			"./tex_image.mha" // output dest
	 			 );




	for (unsigned int j = 0; j < NUM_IMAGES; j++) {
		gpuErrchk( hipFree( dev_pointers_in[j] ) );
		gpuErrchk( hipFree( dev_pointers_out[j] ) );
		hipStreamDestroy( streams_fft[j] );
		gpuErrchk( hipFreeArray( d_volumeArray ) );
		delete[] image_vector[j];
		delete[] mult_image_vector[j];
	}
	gpuErrchk( hipFree( dev_OTF ) );
	delete[] OTF;
	delete[] out;
	delete[] x;
	delete[] y;

	hipDeviceReset();

	// hipfftPlan2d(&planr2c, NY, NX, HIPFFT_D2Z);
	// hipfftPlan2d(&planc2r, NY, NX, HIPFFT_Z2D);

	// hipfftExecD2Z(planr2c, (hipfftDoubleReal *)d_vx, (hipfftDoubleComplex *)d_out);
	// hipfftExecZ2D(planc2r, (hipfftDoubleComplex *)d_out, (hipfftDoubleReal *)d_vx);


	/* Copy results back from the device */
	// hipMemcpy(vx, d_vx, NX * NY * sizeof(hipfftDoubleReal), hipMemcpyDeviceToHost);


    // int count = 0;
    // hipDeviceProp_t prop;
    // int dev_id;
    // //  determining how many devices are available to use on the computer
    // HANDLE_ERROR( hipGetDeviceCount( &count ) );
    // printf("There are %d device(s) on this computer.\n", count);
    // // Iterates through each of the device on this computer

    // printDevInfo(count, prop);



	return 0;
}
#pragma clang diagnostic pop